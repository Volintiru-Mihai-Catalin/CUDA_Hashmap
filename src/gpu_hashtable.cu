#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

__device__ unsigned int transf_key_to_hash(int key, int sizeTable) {
	unsigned int hash = (unsigned int) key;
	
	hash ^= hash >> 16;
	hash *= K;
    hash ^= hash >> 13;
    hash *= M;
	hash ^= hash >> 16;

	return hash % sizeTable;
}

__device__ void insert_element(unsigned int starting_hash, int key, int value, hash_T table, int table_size) {
	unsigned int hash_key = starting_hash % table_size;
	bool inserted = false;

	while (!inserted) {
		int old_value = atomicCAS(&table[hash_key].key, 0, key);
		if (old_value == 0 || old_value == key) {
			table[hash_key].value = value;
			inserted = true;
		}
		hash_key++;
		hash_key = hash_key % table_size;
	}
}

__global__ void batch_insert(int *keys, int *values, int numKeys, hash_T table, int table_size) {
	
	unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < numKeys) {
		unsigned int hash_key = transf_key_to_hash(keys[index], table_size);
		insert_element(hash_key, keys[index], values[index], table, table_size);
	}

}

__global__ void transfer_elements(hash_T new_hash_map, int new_size, hash_T old_hash_map, int old_size) {

	unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;

	if (index < old_size && old_hash_map[index].key != 0) {
		unsigned int new_hash = transf_key_to_hash(old_hash_map[index].key, new_size);
		insert_element(new_hash, old_hash_map[index].key, old_hash_map[index].value, new_hash_map, new_size);
	}
}

__global__ void batch_find(int *keys, int *values, int numKeys, hash_T table, int table_size) {
	unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
	bool found = false;

	if (index < numKeys) {
		unsigned int hash_key = transf_key_to_hash(keys[index], table_size);
		while (!found) {
			if (table[hash_key].key == keys[index]) {
				values[index] = table[hash_key].value;
				found = true;
			}
			hash_key++;
			hash_key = hash_key % table_size;
		}
	}
}
/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
	
	glbGpuAllocator->_cudaMalloc((void **) &this->hash_table, sizeof(hash_t) * size);
	hipMemset(this->hash_table, 0, sizeof(hash_t) * size);
	
	this->hash_table_size = size;
	this->num_elem = 0;

	this->block_size = BLOCK_SIZE;
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree(this->hash_table);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	
	hash_T new_hash_table = 0;
	size_t block_no = hash_table_size / block_size;

	if (hash_table_size % block_size) {
		block_no++;
	}

	glbGpuAllocator->_cudaMalloc((void **) &new_hash_table, sizeof(hash_t) * numBucketsReshape);
	hipMemset(new_hash_table, 0, sizeof(hash_t) * numBucketsReshape);

	transfer_elements<<<block_no, block_size>>>(new_hash_table, numBucketsReshape, hash_table, hash_table_size);
	hipDeviceSynchronize();


	glbGpuAllocator->_cudaFree(this->hash_table);
	this->hash_table = new_hash_table;
	hash_table_size = numBucketsReshape;

	return;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {

	int *device_keys = 0;
	int *device_values = 0;
	size_t block_no = numKeys / block_size;
	
	if (numKeys % block_size) {
		block_no++;
	}
	
	glbGpuAllocator->_cudaMalloc((void **) &device_keys, sizeof(int) * numKeys);
	glbGpuAllocator->_cudaMalloc((void **) &device_values, sizeof(int) * numKeys);
	
	hipMemcpy(device_keys, keys, sizeof(int) * numKeys, hipMemcpyHostToDevice);
	hipMemcpy(device_values, values, sizeof(int) * numKeys, hipMemcpyHostToDevice);


	if ((num_elem + numKeys) / float(hash_table_size) > LOAD_FACTOR_HIGH) {
		reshape((int) ((num_elem + numKeys) / LOAD_FACTOR_LOW));
	}

	batch_insert<<<block_no, block_size>>>(device_keys, device_values, numKeys, this->hash_table, this->hash_table_size);
	hipDeviceSynchronize();

	num_elem += numKeys;

	glbGpuAllocator->_cudaFree(device_keys);
	glbGpuAllocator->_cudaFree(device_values);

	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {

	int *host_values = 0;
	int *device_values = 0;
	int *device_keys = 0;
	size_t block_no = numKeys / block_size;

	if (numKeys % block_size) {
		block_no++;
	}
	
	glbGpuAllocator->_cudaMalloc((void **) &device_keys, sizeof(int) * numKeys);
	hipMemcpy(device_keys, keys, sizeof(int) * numKeys, hipMemcpyHostToDevice);

	glbGpuAllocator->_cudaMalloc((void **) &device_values, sizeof(int) * numKeys);
	host_values = (int *)malloc(sizeof(int) * numKeys);

	batch_find<<<block_no, block_size>>>(device_keys, device_values, numKeys, this->hash_table, this->hash_table_size);
	hipDeviceSynchronize();

	hipMemcpy(host_values, device_values, sizeof(int) * numKeys, hipMemcpyDeviceToHost);

	glbGpuAllocator->_cudaFree(device_keys);
	glbGpuAllocator->_cudaFree(device_values);

	return host_values;
}
